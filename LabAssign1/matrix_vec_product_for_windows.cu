#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <hip/hip_runtime.h>
#include <stdio.h>

//You can change the dimension, program will produce two matrices.
#define M 10
#define N 10

#define CUDA_CALL(x) {if((x) != hipSuccess){ \
  printf("CUDA error at %s:%d\n",__FILE__,__LINE__); \
  printf("  %s\n", hipGetErrorString(hipGetLastError())); \
  exit(EXIT_FAILURE);}}

__global__ void matrixAdd(int d_x[][N], int d_y[N], int d_z[][N]) {
	int idx = threadIdx.x;
	int idy = threadIdx.y;
	if (idx < M && idy < N) {
		d_z[idx][idy] = d_x[idx][idy] * d_y[idy];
	}
}

int main() {

	int sizeM = (M * N) * sizeof(int);
	int sizeV = N * sizeof(int);

	int h_x[M][N], h_y[N], h_z[M][N];
	int(*d_x)[N], (*d_y), (*d_z)[N];

	int i = 0;
	int j = 0;

	//Initialize matrix
	for (i = 0; i < M; i++) {
		for (int j = 0; j < N; j++) {
			h_x[i][j] = 1;
			h_z[i][j] = 0;
		}
	}

	//Initialize vector
	for (i = 0; i < N; i++) {
		h_y[i] = 2;
	}


	hipEvent_t startC, stopC;
	float elapsed_time_msC;
	hipEventCreate(&startC);
	hipEventCreate(&stopC);
	hipEventRecord(startC, 0);
	for (i = 0; i < M; i++) {
		for (int j = 0; j < N; j++) {
			h_z[i][j] = h_x[i][j] * h_y[j];
		}
	}
	hipEventRecord(stopC, 0);
	hipEventSynchronize(stopC);
	hipEventElapsedTime(&elapsed_time_msC, startC, stopC);
	printf("Time to calculate results(CPU Time): %f ms.\n", elapsed_time_msC);

	CUDA_CALL(hipMalloc(&d_x, sizeM));
	CUDA_CALL(hipMemcpy(d_x, h_x, sizeM, hipMemcpyHostToDevice));

	CUDA_CALL(hipMalloc(&d_y, sizeV));
	CUDA_CALL(hipMemcpy(d_y, h_y, sizeV, hipMemcpyHostToDevice));

	CUDA_CALL(hipMalloc(&d_z, sizeM));

	dim3 dimGrid(1, 1);
	dim3 dimBlock(M, N);

	hipEvent_t start, stop;
	float elapsed_time_ms;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, 0);

	matrixAdd << < dimGrid, dimBlock >> > (d_x, d_y, d_z);
	CUDA_CALL(hipMemcpy(h_z, d_z, sizeM, hipMemcpyDeviceToHost));

	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&elapsed_time_ms, start, stop);
	printf("Time to calculate results(GPU Time): %f ms.\n", elapsed_time_ms);

	hipFree(d_x);
	hipFree(d_y);
	hipFree(d_z);

	printf("Output of Multiplication\n");
	for (i = 0; i < M; i++) {
		for (j = 0; j < N; j++) {
			printf("%d\t", h_z[i][j]);
		}
		printf("\n");
	}
	printf("\n");
	getchar();
}