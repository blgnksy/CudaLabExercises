#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#define CUDA_CALL(x) {if((x) != hipSuccess){ \
  printf("CUDA error at %s:%d\n",__FILE__,__LINE__); \
  printf("  %s\n", hipGetErrorString(hipGetLastError())); \
  exit(EXIT_FAILURE);}}

__global__ void MatrixMulKernel(float *d_x, float *d_y, float *d_z, int Width) {

    int idx = threadIdx.x;
    int idy = threadIdx.y;

    float kernelSum = 0;
    if ((idx < Width) && (idy < Width)) {
        for (int k = 0; k < Width; ++k) {
            kernelSum += d_x[idy * Width + k] * d_y[k * Width + idx];
        }
        d_z[idy * Width + idx] = kernelSum;
    }
}

int main(void) {

    int Width = 60;
    int size = Width * Width * sizeof(float);
    float h_x[Width * Width], h_y[Width * Width], h_z[Width * Width];
    float *d_x, *d_y, *d_z;

    for (int i = 0; i < (Width * Width); i++) {
        h_x[i] = i;
        h_y[i] = i;
        h_z[i] = 0;
    }

    hipEvent_t start, stop;
    float elapsed_time_ms;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);


    CUDA_CALL(hipMalloc((void **) &d_x, size));
    CUDA_CALL(hipMemcpy(d_x, h_x, size, hipMemcpyHostToDevice));
    CUDA_CALL(hipMalloc((void **) &d_y, size));
    CUDA_CALL(hipMemcpy(d_y, h_y, size, hipMemcpyHostToDevice));
    CUDA_CALL(hipMalloc((void **) &d_z, size));


    dim3 dimBlock(Width, Width);
    dim3 dimGrid(1, 1);

    MatrixMulKernel << < dimGrid, dimBlock >> > (d_x, d_y, d_z, Width);

    CUDA_CALL(hipMemcpy(h_z, d_z, size, hipMemcpyDeviceToHost));

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsed_time_ms, start, stop);
    printf("Time to calculate results(GPU Time): %f ms.\n", elapsed_time_ms);

    //for (int i = 0; i < (Width * Width); i++) {
    //    printf("%0.2f \n", h_z[i]);
    //}

    hipFree(d_x);
    hipFree(d_y);
    hipFree(d_z);
}
