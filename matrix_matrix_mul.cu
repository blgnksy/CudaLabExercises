#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>




#define CUDA_CALL(x) {if((x) != hipSuccess){ \
  printf("CUDA error at %s:%d\n",__FILE__,__LINE__); \
  printf("  %s\n", hipGetErrorString(hipGetLastError())); \
  exit(EXIT_FAILURE);}}

__global__ void MatrixMulKernel(float *d_x, float *d_y, float *d_z, int Width) {

    int idx = threadIdx.x;
    int idy = threadIdx.y;

    float kernelSum = 0;

    for(int k = 0; k < Width ; ++k) {
        kernelSum += d_x[idy*Width + k]*d_y[k*Width + idx];
    }

    d_z[idy*Width + idx] = kernelSum;
}

int main() {

    int Width=5;
    int size = Width*Width*sizeof(float);
    float h_x[Width*Width], h_y[Width*Width], h_z[Width*Width];
    float *d_x, *d_y, *d_z;

    for(int i = 0; i < (Width*Width) ; i++) {
        h_x[i] = i;
        h_y[i] = i;
        h_z[i] = 0;
    }




    hipEvent_t start, stop;
    float elapsed_time_ms;
    hipEventCreate( &start );
    hipEventCreate( &stop );
    hipEventRecord( start, 0 );


    hipMalloc((void**)&d_x, size);
    hipMemcpy(d_x,h_x,size,hipMemcpyHostToDevice);
    hipMalloc((void**)&d_y, size);
    hipMemcpy(d_y,h_y,size,hipMemcpyHostToDevice);
    hipMalloc((void**)&d_z,size);


    dim3 dimBlock(Width,Width);
    dim3 dimGrid(1,1);

    MatrixMulKernel<<<dimGrid,dimBlock>>>(d_x,d_y,d_z,Width);

    hipMemcpy(h_z,d_z,size,hipMemcpyDeviceToHost);

    hipEventRecord( stop, 0 );
    hipEventSynchronize( stop );
    hipEventElapsedTime( &elapsed_time_ms, start, stop );
    printf("Time to calculate results(GPU Time): %f ms.\n", elapsed_time_ms);

    for(int i = 0; i < (Width*Width) ; i++) {
        printf("%0.2f \n", h_z[i]);
    }

    hipFree(d_x);
    hipFree(d_y);
    hipFree(d_z);
}
