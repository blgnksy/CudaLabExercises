#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>



__global__ void vecProduct(int *d_x, int *d_y, int *d_z, int N) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < N) {
        d_z[idx] = d_x[idx] * d_y[idx];
    }
}

int main() {
    int N=0;

    printf("%s","Enter the size of vector : ");

    if( scanf( "%d", &N) == 0 )
    {
        fprintf( stderr, "Expected a positive number as input\n");
        exit(1);
    }
    int size = N * sizeof(int);
    int h_x[N], h_y[N], h_z[N], *d_x, *d_y, *d_z;
    int i = 0;

    int total=0;

    //Initialize vectors
    for (i = 0; i < N; i++) {
        h_x[i] = i;
        h_y[i] = i;
        h_z[i] = 0;
    }

    hipEvent_t startC, stopC;
    float elapsed_time_msC;
    hipEventCreate( &startC );
    hipEventCreate( &stopC );
    hipEventRecord( startC, 0 );
    for (i = 0; i < N; i++) {
        h_z[i] =h_x[i]+h_y[i] ;
    }
    hipEventRecord( stopC, 0 );
    hipEventSynchronize( stopC );
    hipEventElapsedTime( &elapsed_time_msC, startC, stopC );
    printf("Time to calculate results(CPU Time): %f ms.\n", elapsed_time_msC);

    hipMalloc(&d_x, size);
    hipMemcpy(d_x, h_x, size, hipMemcpyHostToDevice);

    hipMalloc(&d_y, size);
    hipMemcpy(d_y, h_y, size, hipMemcpyHostToDevice);

    hipMalloc(&d_z, size);

    dim3 dimGrid(1, 1);
    dim3 dimBlock(N, 1);

    hipEvent_t start, stop;
    float elapsed_time_ms;
    hipEventCreate( &start );
    hipEventCreate( &stop );
    hipEventRecord( start, 0 );

    vecProduct <<<  dimGrid, dimBlock >>> (d_x, d_y, d_z, N);
    hipMemcpy(h_z, d_z, size, hipMemcpyDeviceToHost);

    hipEventRecord( stop, 0 );
    hipEventSynchronize( stop );
    hipEventElapsedTime( &elapsed_time_ms, start, stop );

    hipFree(d_x);
    hipFree(d_y);
    hipFree(d_z);

    for (i = 0; i < N; i++) {
        total+= h_z[i];
    }
    printf("Time to calculate results(GPU Time): %f ms.\n", elapsed_time_ms);
}
