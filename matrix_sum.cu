#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>


//You can change the dimension, program will produce two matrices.
#define M 600
#define N 800

#define CUDA_CALL(x) {if((x) != hipSuccess){ \
  printf("CUDA error at %s:%d\n",__FILE__,__LINE__); \
  printf("  %s\n", hipGetErrorString(hipGetLastError())); \
  exit(EXIT_FAILURE);}}

__global__ void matrixAdd(int d_x[][N], int d_y[][N], int d_z[][N]) {
    int idx = threadIdx.x;
    int idy = threadIdx.y;
    if (idx < M && idy < N) {
        d_z[idx][idy] = d_x[idx][idy] + d_y[idx][idy];
    }
}

int main() {

    int size = (M * N) * sizeof(int);

    int h_x[M][N], h_y[M][N], h_z[M][N];
    int(*d_x)[N], (*d_y)[N], (*d_z)[N];

    int i = 0;
    int j = 0;

    //Initialize matrix
    for (i = 0; i < M; i++) {
        for (int j = 0; j < N; j++) {
            h_x[i][j] = M;
            h_y[i][j] = N;
            h_z[i][j] = 0;
        }
    }

    hipEvent_t startC, stopC;
    float elapsed_time_msC;
    hipEventCreate( &startC );
    hipEventCreate( &stopC );
    hipEventRecord( startC, 0 );
    for (i = 0; i < M; i++) {
        for (int j = 0; j < N; j++) {
            h_z[i][j] =h_x[i][j] + h_y[i][j] ;
        }
    }
    hipEventRecord( stopC, 0 );
    hipEventSynchronize( stopC );
    hipEventElapsedTime( &elapsed_time_msC, startC, stopC );
    printf("Time to calculate results(CPU Time): %f ms.\n", elapsed_time_msC);

    CUDA_CALL(hipMalloc(&d_x, size));
    CUDA_CALL(hipMemcpy(d_x, h_x, size, hipMemcpyHostToDevice));

    CUDA_CALL(hipMalloc(&d_y, size));
    CUDA_CALL(hipMemcpy(d_y, h_y, size, hipMemcpyHostToDevice));

    CUDA_CALL(hipMalloc(&d_z, size));


    dim3 dimGrid(1, 1);
    dim3 dimBlock(M, N);

    hipEvent_t start, stop;
    float elapsed_time_ms;
    hipEventCreate( &start );
    hipEventCreate( &stop );
    hipEventRecord( start, 0 );

    matrixAdd <<< dimGrid, dimBlock >>> (d_x, d_y, d_z);
    CUDA_CALL(hipMemcpy(h_z, d_z, size, hipMemcpyDeviceToHost));

    hipEventRecord( stop, 0 );
    hipEventSynchronize( stop );
    hipEventElapsedTime( &elapsed_time_ms, start, stop );
    printf("Time to calculate results(GPU Time): %f ms.\n", elapsed_time_ms);

    hipFree(d_x);
    hipFree(d_y);
    hipFree(d_z);

    printf("Output of Summation\n");
//    for (i = 0; i<M; i++) {
//        for (j = 0; j<N; j++) {
//            printf("%d\t", h_z[i][j]);
//        }
//        printf("\n");
//    }
    printf("\n");
}
